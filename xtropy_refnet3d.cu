
#include <hip/hip_runtime.h>

__device__ int get_index_x (int ncols, int index ) { 
  if (index == -1)  {
    index = blockDim.x * blockIdx.x + threadIdx.x;
  } else {
    index += gridDim.x;
  }

  if (index >= ncols) index = -1;

  return index;
}

__device__ int get_index_y (int nrows, int index ) { 
  if (index == -1)  {
    index = blockDim.y * blockIdx.y + threadIdx.y;
  } else {
    index += gridDim.y;
  }

  if (index >= nrows) index = -1;

  return index;
}

__device__ int get_index_z (int depth, int index ) { 
  if (index == -1)  {
    index = blockDim.z * blockIdx.z + threadIdx.z;
  } else {
    index += gridDim.z;
  }

  if (index >= depth) index = -1;

  return index;
}

__device__ float Gaussian (float x, float y, float sigma) {
  return (1/sqrt(2*M_PI*sigma)) * exp((-pow(x - y,2)) / (2*pow(sigma,2)));
}

__device__ float Gaussian_prime (float x, float y, float sigma) {
  return ( -(x - y) / ((pow(sigma,3))*sqrt(2*M_PI)) ) *  exp((-pow(x - y,2)) / (2*pow(sigma,2)));
}


__global__ void ACm( float *out, const float *x, const float *y, const float *wx, const float *wy,  const int *marray, float sigma, int msize, int ncols, int nrows, int depth) {
  float sum = 0;
  int i = 0;
  int idm = get_index_x(msize, -1);
  int idy = get_index_y(nrows, -1);
  int idz = get_index_z(depth, -1);
  int m = marray[idm];
  int cn = 1;

  int li;
  int lm;

  while(idz >= 0) {
    while (idy >= 0) {
      while(idm >= 0) {
        sum = 0;
        cn = 1;

        li = max(0,m);
        if (m < 0)
          lm = ncols + m ;
        else
          lm = ncols;


        for (i=li; i < lm; i++) {

          sum += Gaussian (x[i + idy*ncols + idz*nrows*ncols] * wx[idm + i*msize], y[abs(i-m) + idy*ncols + idz*nrows*ncols] * wy[idm + abs(i-m)*msize], sigma);

          cn = cn + 1;
        }

        out[idm + idy*msize + idz*nrows*msize] = ( 1/((float)cn) ) * sum;

        idm = get_index_x(msize, idm);
        m = marray[idm];

      }
      idy = get_index_y (nrows, idy);
    }
    idz = get_index_z (depth, idz);
  }
}

__global__ void ACm_prime( float *out, const float *x, const float *y, const float *wx, const float *wy,  const int *marray, float sigma, int msize, int ncols, int nrows, int depth) {
  float sum = 0;
  int i = 0;
  int idm = get_index_x(msize, -1);
  int idy = get_index_y(nrows, -1);
  int idz = get_index_z(depth, -1);
  int m = marray[idm];
  int cn = 1;

  int li;
  int lm;

  while(idz >= 0) {
    while (idy >= 0) {
      while(idm >= 0) {
        sum = 0;
        cn = 1;

        li = max(0,m);
        if (m < 0)
          lm = ncols + m ;
        else
          lm = ncols;

        for (i=li; i < lm; i++) {

          sum += Gaussian_prime (x[i + idy*ncols + idz*nrows*ncols] * wx[idm + i*msize], y[abs(i-m) + idy*ncols + idz*nrows*ncols] * wy[idm + abs(i-m)*msize], sigma);

          cn = cn + 1;
        }

        out[idm + idy*msize + idz*nrows*msize] = ( 1/((float)cn) ) * sum;

        idm = get_index_x(msize, idm);
        m = marray[idm];

      }
      idy = get_index_y (nrows, idy);
    }
    idz = get_index_z (depth, idz);
  }
}
